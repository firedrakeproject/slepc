#include "hip/hip_runtime.h"
/*
   - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
   SLEPc - Scalable Library for Eigenvalue Problem Computations
   Copyright (c) 2002-, Universitat Politecnica de Valencia, Spain

   This file is part of SLEPc.
   SLEPc is distributed under a 2-clause BSD license (see LICENSE).
   - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
*/
/*
   BV orthogonalization routines (CUDA)
*/

#include <slepc/private/bvimpl.h>          /*I   "slepcbv.h"   I*/
#include <slepcblaslapack.h>
#include <slepccublas.h>

/*
   BV_CleanCoefficients_CUDA - Sets to zero all entries of column j of the bv buffer
*/
PetscErrorCode BV_CleanCoefficients_CUDA(BV bv,PetscInt j,PetscScalar *h)
{
  PetscScalar    *d_hh,*d_a;
  PetscInt       i;

  PetscFunctionBegin;
  if (!h) {
    PetscCall(VecCUDAGetArray(bv->buffer,&d_a));
    PetscCall(PetscLogGpuTimeBegin());
    d_hh = d_a + j*(bv->nc+bv->m);
    PetscCallCUDA(hipMemset(d_hh,0,(bv->nc+j)*sizeof(PetscScalar)));
    PetscCall(PetscLogGpuTimeEnd());
    PetscCall(VecCUDARestoreArray(bv->buffer,&d_a));
  } else { /* cpu memory */
    for (i=0;i<bv->nc+j;i++) h[i] = 0.0;
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/*
   BV_AddCoefficients_CUDA - Add the contents of the scratch (0-th column) of the bv buffer
   into column j of the bv buffer
 */
PetscErrorCode BV_AddCoefficients_CUDA(BV bv,PetscInt j,PetscScalar *h,PetscScalar *c)
{
  PetscScalar    *d_h,*d_c,sone=1.0;
  PetscInt       i;
  PetscCuBLASInt idx=0,one=1;
  hipblasHandle_t cublasv2handle;

  PetscFunctionBegin;
  if (!h) {
    PetscCall(PetscCUBLASGetHandle(&cublasv2handle));
    PetscCall(VecCUDAGetArray(bv->buffer,&d_c));
    d_h = d_c + j*(bv->nc+bv->m);
    PetscCall(PetscCuBLASIntCast(bv->nc+j,&idx));
    PetscCall(PetscLogGpuTimeBegin());
    PetscCallCUBLAS(cublasXaxpy(cublasv2handle,idx,&sone,d_c,one,d_h,one));
    PetscCall(PetscLogGpuTimeEnd());
    PetscCall(PetscLogGpuFlops(1.0*(bv->nc+j)));
    PetscCall(VecCUDARestoreArray(bv->buffer,&d_c));
  } else { /* cpu memory */
    for (i=0;i<bv->nc+j;i++) h[i] += c[i];
    PetscCall(PetscLogFlops(1.0*(bv->nc+j)));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/*
   BV_SetValue_CUDA - Sets value in row j (counted after the constraints) of column k
   of the coefficients array
*/
PetscErrorCode BV_SetValue_CUDA(BV bv,PetscInt j,PetscInt k,PetscScalar *h,PetscScalar value)
{
  PetscScalar    *d_h,*a;

  PetscFunctionBegin;
  if (!h) {
    PetscCall(VecCUDAGetArray(bv->buffer,&a));
    PetscCall(PetscLogGpuTimeBegin());
    d_h = a + k*(bv->nc+bv->m) + bv->nc+j;
    PetscCallCUDA(hipMemcpy(d_h,&value,sizeof(PetscScalar),hipMemcpyHostToDevice));
    PetscCall(PetscLogCpuToGpu(sizeof(PetscScalar)));
    PetscCall(PetscLogGpuTimeEnd());
    PetscCall(VecCUDARestoreArray(bv->buffer,&a));
  } else { /* cpu memory */
    h[bv->nc+j] = value;
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/*
   BV_SquareSum_CUDA - Returns the value h'*h, where h represents the contents of the
   coefficients array (up to position j)
*/
PetscErrorCode BV_SquareSum_CUDA(BV bv,PetscInt j,PetscScalar *h,PetscReal *sum)
{
  const PetscScalar *d_h;
  PetscScalar       dot;
  PetscInt          i;
  PetscCuBLASInt    idx=0,one=1;
  hipblasHandle_t    cublasv2handle;

  PetscFunctionBegin;
  if (!h) {
    PetscCall(PetscCUBLASGetHandle(&cublasv2handle));
    PetscCall(VecCUDAGetArrayRead(bv->buffer,&d_h));
    PetscCall(PetscCuBLASIntCast(bv->nc+j,&idx));
    PetscCall(PetscLogGpuTimeBegin());
    PetscCallCUBLAS(cublasXdotc(cublasv2handle,idx,d_h,one,d_h,one,&dot));
    PetscCall(PetscLogGpuTimeEnd());
    PetscCall(PetscLogGpuFlops(2.0*(bv->nc+j)));
    *sum = PetscRealPart(dot);
    PetscCall(VecCUDARestoreArrayRead(bv->buffer,&d_h));
  } else { /* cpu memory */
    *sum = 0.0;
    for (i=0;i<bv->nc+j;i++) *sum += PetscRealPart(h[i]*PetscConj(h[i]));
    PetscCall(PetscLogFlops(2.0*(bv->nc+j)));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* pointwise multiplication */
static __global__ void PointwiseMult_kernel(PetscInt xcount,PetscScalar *a,const PetscScalar *b,PetscInt n)
{
  PetscInt x;

  x = xcount*gridDim.x*blockDim.x+blockIdx.x*blockDim.x+threadIdx.x;
  if (x<n) a[x] *= PetscRealPart(b[x]);
}

/* pointwise division */
static __global__ void PointwiseDiv_kernel(PetscInt xcount,PetscScalar *a,const PetscScalar *b,PetscInt n)
{
  PetscInt x;

  x = xcount*gridDim.x*blockDim.x+blockIdx.x*blockDim.x+threadIdx.x;
  if (x<n) a[x] /= PetscRealPart(b[x]);
}

/*
   BV_ApplySignature_CUDA - Computes the pointwise product h*omega, where h represents
   the contents of the coefficients array (up to position j) and omega is the signature;
   if inverse=TRUE then the operation is h/omega
*/
PetscErrorCode BV_ApplySignature_CUDA(BV bv,PetscInt j,PetscScalar *h,PetscBool inverse)
{
  PetscScalar       *d_h;
  const PetscScalar *d_omega,*omega;
  PetscInt          i,xcount;
  dim3              blocks3d, threads3d;

  PetscFunctionBegin;
  if (!(bv->nc+j)) PetscFunctionReturn(PETSC_SUCCESS);
  if (!h) {
    PetscCall(VecCUDAGetArray(bv->buffer,&d_h));
    PetscCall(VecCUDAGetArrayRead(bv->omega,&d_omega));
    PetscCall(SlepcKernelSetGrid1D(bv->nc+j,&blocks3d,&threads3d,&xcount));
    PetscCall(PetscLogGpuTimeBegin());
    if (inverse) {
      for (i=0;i<xcount;i++) PointwiseDiv_kernel<<<blocks3d,threads3d>>>(i,d_h,d_omega,bv->nc+j);
    } else {
      for (i=0;i<xcount;i++) PointwiseMult_kernel<<<blocks3d,threads3d>>>(i,d_h,d_omega,bv->nc+j);
    }
    PetscCallCUDA(hipGetLastError());
    PetscCall(PetscLogGpuTimeEnd());
    PetscCall(PetscLogGpuFlops(1.0*(bv->nc+j)));
    PetscCall(VecCUDARestoreArrayRead(bv->omega,&d_omega));
    PetscCall(VecCUDARestoreArray(bv->buffer,&d_h));
  } else {
    PetscCall(VecGetArrayRead(bv->omega,&omega));
    if (inverse) for (i=0;i<bv->nc+j;i++) h[i] /= PetscRealPart(omega[i]);
    else for (i=0;i<bv->nc+j;i++) h[i] *= PetscRealPart(omega[i]);
    PetscCall(VecRestoreArrayRead(bv->omega,&omega));
    PetscCall(PetscLogFlops(1.0*(bv->nc+j)));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/*
   BV_SquareRoot_CUDA - Returns the square root of position j (counted after the constraints)
   of the coefficients array
*/
PetscErrorCode BV_SquareRoot_CUDA(BV bv,PetscInt j,PetscScalar *h,PetscReal *beta)
{
  const PetscScalar *d_h;
  PetscScalar       hh;

  PetscFunctionBegin;
  if (!h) {
    PetscCall(VecCUDAGetArrayRead(bv->buffer,&d_h));
    PetscCall(PetscLogGpuTimeBegin());
    PetscCallCUDA(hipMemcpy(&hh,d_h+bv->nc+j,sizeof(PetscScalar),hipMemcpyDeviceToHost));
    PetscCall(PetscLogGpuToCpu(sizeof(PetscScalar)));
    PetscCall(PetscLogGpuTimeEnd());
    PetscCall(BV_SafeSqrt(bv,hh,beta));
    PetscCall(VecCUDARestoreArrayRead(bv->buffer,&d_h));
  } else PetscCall(BV_SafeSqrt(bv,h[bv->nc+j],beta));
  PetscFunctionReturn(PETSC_SUCCESS);
}

/*
   BV_StoreCoefficients_CUDA - Copy the contents of the coefficients array to an array dest
   provided by the caller (only values from l to j are copied)
*/
PetscErrorCode BV_StoreCoefficients_CUDA(BV bv,PetscInt j,PetscScalar *h,PetscScalar *dest)
{
  const PetscScalar *d_h,*d_a;
  PetscInt          i;

  PetscFunctionBegin;
  if (!h) {
    PetscCall(VecCUDAGetArrayRead(bv->buffer,&d_a));
    PetscCall(PetscLogGpuTimeBegin());
    d_h = d_a + j*(bv->nc+bv->m)+bv->nc;
    PetscCallCUDA(hipMemcpy(dest-bv->l,d_h,(j-bv->l)*sizeof(PetscScalar),hipMemcpyDeviceToHost));
    PetscCall(PetscLogGpuToCpu((j-bv->l)*sizeof(PetscScalar)));
    PetscCall(PetscLogGpuTimeEnd());
    PetscCall(VecCUDARestoreArrayRead(bv->buffer,&d_a));
  } else {
    for (i=bv->l;i<j;i++) dest[i-bv->l] = h[bv->nc+i];
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}
